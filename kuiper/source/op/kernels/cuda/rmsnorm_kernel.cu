#include "hip/hip_runtime.h"
#include <cub/block/block_reduce.cuh>
#include "rmsnorm_kernel.cuh"
namespace kernel {
template <int32_t BLOCK_DIM>
static __global__ void row_rmsnorm_f32(float* in, float* wei, float* out, int size, float eps) {
  const int tid = threadIdx.x;

  constexpr int pack_size = 4;
  const int pack_num = size / pack_size;
  const int pack_off = pack_size * pack_num;

  float sum = 0.0f;
  float4* in_pack = reinterpret_cast<float4*>(in);
  for (int i = tid; i < pack_num; i += blockDim.x) {
    float4 in_float4 = *(in_pack + i);
    sum += in_float4.x * in_float4.x;
    sum += in_float4.y * in_float4.y;
    sum += in_float4.z * in_float4.z;
    sum += in_float4.w * in_float4.w;
  }

  for (int i = pack_off + tid; i < size; i += blockDim.x) {
    sum += in[i] * in[i];
  }

  using BlockReduce = hipcub::BlockReduce<float, BLOCK_DIM>;
  __shared__ typename BlockReduce::TempStorage temp;
  __shared__ float shared_val;
  sum = BlockReduce(temp).Sum(sum);
  if (threadIdx.x == 0) {
    shared_val = sum;
  }
  __syncthreads();
  sum = shared_val;
  const float scale = rsqrtf(sum / static_cast<float>(size) + eps);

  float4* wei_pack = reinterpret_cast<float4*>(wei);
  float4* out_pack = reinterpret_cast<float4*>(out);
  for (int i = tid; i < pack_num; i += blockDim.x) {
    float4 in_float4 = *(in_pack + i);
    float4 wei_float4 = *(wei_pack + i);
    *(out_pack + i) =
        make_float4(scale * in_float4.x * wei_float4.x, scale * in_float4.y * wei_float4.y,
                    scale * in_float4.z * wei_float4.z, scale * in_float4.w * wei_float4.w);
  }

  for (int i = pack_off + tid; i < size; i += blockDim.x) {
    out[i] = wei[i] * in[i] * scale;
  }
}

void rmsnorm_kernel_cu(const tensor::Tensor& input, const tensor::Tensor& weight,
                       const tensor::Tensor& output, void* stream) {
  CHECK(!input.is_empty());
  CHECK(!weight.is_empty());
  CHECK(!output.is_empty());

  CHECK(input.device_type() == base::DeviceType::kDeviceCUDA &&
        weight.device_type() == base::DeviceType::kDeviceCUDA &&
        output.device_type() == base::DeviceType::kDeviceCUDA);

  const float eps = 1e-5f;
  const int32_t size = static_cast<int32_t>(input.size());
  float* in_ptr = const_cast<float*>(input.ptr<float>());
  float* wei_ptr = const_cast<float*>(weight.ptr<float>());
  float* out_ptr = const_cast<float*>(output.ptr<float>());
  constexpr int threads_num = 128;
  if (stream) {
    hipStream_t stream_ = static_cast<hipStream_t>(stream);
    row_rmsnorm_f32<128><<<1, threads_num, 0, stream_>>>(in_ptr, wei_ptr, out_ptr, size, eps);
  } else {
    row_rmsnorm_f32<128><<<1, threads_num>>>(in_ptr, wei_ptr, out_ptr, size, eps);
  }
}
}  // namespace kernel