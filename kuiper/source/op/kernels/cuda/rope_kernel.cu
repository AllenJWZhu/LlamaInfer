#include "hip/hip_runtime.h"
#include "rope_kernel.cuh"
namespace kernel {

__device__ void rope_calc(float fcr, float fci, float* vec, int32_t idx) {
  float2* vec_ptr = reinterpret_cast<float2*>(vec + idx);
  float2 vec_value = *vec_ptr;
  *vec_ptr =
      make_float2(vec_value.x * fcr - vec_value.y * fci, vec_value.x * fci + vec_value.y * fcr);
}

__global__ void rope_kernel_cu_fp32(int pos, int dim, int kv_dim, int head_size,
                                    const float* input_q, const float* input_k,
                                    const float* sin_cache, const float* cos_cache) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  idx = idx * 2;
  if (idx >= dim) {
    return;
  }

  int head_dim = idx % head_size;
  float fci = *(sin_cache + pos * head_size + head_dim);
  float fcr = *(cos_cache + pos * head_size + head_dim);

  rope_calc(fcr, fci, const_cast<float*>(input_q), idx);
  if (idx >= kv_dim) {
    return;
  }
  rope_calc(fcr, fci, const_cast<float*>(input_k), idx);
}

__global__ void sin_cos_calc(int head_size, int max_seq_len, float* sin_cache, float* cos_cache) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int head_dim = idx % head_size;
  for (int pos = 0; pos < max_seq_len; ++pos) {
    float freq = 1.0f / pow(10000.0f, static_cast<float>(head_dim) / static_cast<float>(head_size));
    float val = static_cast<float>(pos) * freq;
    float fcr = cosf(val);
    float fci = sinf(val);
    *(sin_cache + pos * head_size + head_dim) = fci;
    *(cos_cache + pos * head_size + head_dim) = fcr;
  }
}

void sin_cos_cache_calc_cu(int head_size, int max_seq_len, const tensor::Tensor& sin_cache,
                           const tensor::Tensor& cos_cache, hipStream_t stream) {
  CHECK_EQ(sin_cache.is_empty(), false);
  CHECK_EQ(cos_cache.is_empty(), false);
  int threads = head_size;
  if (stream) {
    sin_cos_calc<<<1, threads, 0, stream>>>(head_size, max_seq_len,
                                            const_cast<float*>(sin_cache.ptr<float>()),
                                            const_cast<float*>(cos_cache.ptr<float>()));
  } else {
    sin_cos_calc<<<1, threads>>>(head_size, max_seq_len, const_cast<float*>(sin_cache.ptr<float>()),
                                 const_cast<float*>(cos_cache.ptr<float>()));
  }
}

void rope_kernel_cu(int32_t dim, int32_t kv_dim, int32_t head_size, const tensor::Tensor& input_q,
                    const tensor::Tensor& input_k, const tensor::Tensor& input_pos,
                    const tensor::Tensor& sin_cache, const tensor::Tensor& cos_cache,
                    void* stream) {
  const int32_t pos = *input_pos.ptr<int32_t>(0);
  int threads = 128;
  int blocks = (dim + threads - 1) / threads;
  if (stream) {
    hipStream_t stream_ = static_cast<hipStream_t>(stream);
    rope_kernel_cu_fp32<<<blocks, threads, 0, stream_>>>(
        pos, dim, kv_dim, head_size, input_q.ptr<float>(), input_k.ptr<float>(),
        sin_cache.ptr<float>(), cos_cache.ptr<float>());
  } else {
    rope_kernel_cu_fp32<<<blocks, threads>>>(pos, dim, kv_dim, head_size, input_q.ptr<float>(),
                                             input_k.ptr<float>(), sin_cache.ptr<float>(),
                                             cos_cache.ptr<float>());
  }
}
}  // namespace kernel