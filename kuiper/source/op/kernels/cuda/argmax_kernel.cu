#include "hip/hip_runtime.h"
#include "../kernels_interface.h"
#include "argmax_kernel.cuh"
#include "tensor/tensor.h"
namespace kernel {
__forceinline__ __device__ void warp_reduce_argmax(float& val, size_t& ptr) {
  float tmp_val;
  size_t tmp_ptr;
  unsigned int mask = __ballot_sync(0xFFFFFFFF, true);
  for (unsigned int k = (warpSize >> 1); k > 0; k >>= 1) {
    tmp_val = __shfl_down_sync(mask, val, k, warpSize);
    tmp_ptr = __shfl_down_sync(mask, ptr, k, warpSize);
    if (ptr == SIZE_MAX || tmp_ptr == SIZE_MAX) continue;
    if (tmp_val > val) {
      val = tmp_val;
      ptr = tmp_ptr;
    } else if (tmp_val == val && tmp_ptr < ptr) {
      ptr = tmp_ptr;
    }
  }
}

__forceinline__ __device__ void block_reduce_argmax(float& val, size_t& ptr, float* shared_value,
                                                    size_t* shared_ptr) {
  int lane_id = threadIdx.x % warpSize;
  int warp_id = threadIdx.x / warpSize;

  warp_reduce_argmax(val, ptr);

  __syncthreads();
  if (lane_id == 0) {
    shared_value[warp_id] = val;
    shared_ptr[warp_id] = ptr;
  }

  __syncthreads();
  if (threadIdx.x < blockDim.x / warpSize) {
    val = shared_value[lane_id];
    ptr = shared_ptr[lane_id];
  } else {
    val = 0;
    ptr = SIZE_MAX;
  }

  if (warp_id == 0) {
    warp_reduce_argmax(val, ptr);
  }
}

__global__ void argmax_kernel_fp32(const float* input_ptr, size_t size, size_t* output_idx) {
  __shared__ size_t shared_max_ptr[32];
  __shared__ float shared_max_value[32];
  uint32_t tid = threadIdx.x;
  if (tid >= size) {
    return;
  }

  size_t max_index = threadIdx.x;
  float max_value = input_ptr[max_index];
  for (size_t i = tid; i < size; i += blockDim.x) {
    if (input_ptr[i] > max_value) {
      max_index = i;
      max_value = input_ptr[i];
    }
  }

  block_reduce_argmax(max_value, max_index, shared_max_value, shared_max_ptr);
  __syncthreads();
  if (threadIdx.x == 0) {
    *output_idx = max_index;
  }
}

size_t argmax_kernel_cu(const float* input_ptr, size_t size, void* stream) {
  std::shared_ptr<base::DeviceAllocator> alloc_cu =
      base::CUDADeviceAllocatorFactory::get_instance();
  size_t* index = static_cast<size_t*>(alloc_cu->allocate(sizeof(size_t)));
  size_t output_index = 0;
  if (!stream) {
    argmax_kernel_fp32<<<1, 512>>>(input_ptr, size, index);
    hipMemcpy(&output_index, index, sizeof(size_t), hipMemcpyDeviceToHost);
  } else {
    hipStream_t stream_ = static_cast<hipStream_t>(stream);
    argmax_kernel_fp32<<<1, 512, 0, stream_>>>(input_ptr, size, index);
    hipMemcpyAsync(&output_index, index, sizeof(size_t), hipMemcpyDeviceToHost, stream_);
  }
  return output_index;
}
}  // namespace kernel