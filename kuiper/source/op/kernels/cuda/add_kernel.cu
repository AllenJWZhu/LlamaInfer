#include "hip/hip_runtime.h"
#include "add_kernel.cuh"

namespace kernel {
__global__ void add_kernel_cu_fp32(int32_t size, const float* in1, const float* in2, float* out) {
  int32_t tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid >= size) {
    return;
  }
  float in_val1 = in1[tid];
  float in_val2 = in2[tid];
  out[tid] = in_val1 + in_val2;
}

void add_kernel_cu(const tensor::Tensor& input1, const tensor::Tensor& input2,
                   const tensor::Tensor& output, void* stream) {
  CHECK_EQ(input1.is_empty(), false);
  CHECK_EQ(input2.is_empty(), false);
  CHECK_EQ(output.is_empty(), false);
  int32_t size = static_cast<int32_t>(input1.size());
  CHECK_EQ(size, input2.size());
  CHECK_EQ(size, output.size());
  int32_t thread_num = 512;
  int32_t block_num = (size + thread_num - 1) / thread_num;
  if (stream) {
    hipStream_t stream_ = static_cast<ihipStream_t*>(stream);
    add_kernel_cu_fp32<<<block_num, thread_num, 0, stream_>>>(
        size, input1.ptr<float>(), input2.ptr<float>(), const_cast<float*>(output.ptr<float>()));
  } else {
    add_kernel_cu_fp32<<<block_num, thread_num>>>(size, input1.ptr<float>(), input2.ptr<float>(),
                                                  const_cast<float*>(output.ptr<float>()));
  }
}
}  // namespace kernel
