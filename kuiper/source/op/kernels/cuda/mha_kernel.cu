#include "hip/hip_runtime.h"
#include <base/cuda_config.h>
#include <tensor/tensor.h>
#include <hipcub/hipcub.hpp>
#include "mha_kernel.cuh"
namespace kernel {
__device__ void softmax_gpu(float* __restrict__ x, int size) {
  int tid = threadIdx.x;
  int step = blockDim.x;

  // find max value (for numerical stability)
  float max_val = tid < size ? x[tid] : 0;
  for (int i = tid + step; i < size; i += step) {
    if (x[i] > max_val) {
      max_val = x[i];
    }
  }
  using BlockReduce = hipcub::BlockReduce<float, 128>;
  __shared__ BlockReduce::TempStorage temp;
  __shared__ float shared_val;
  max_val = BlockReduce(temp).Reduce(max_val, hipcub::Max());
  if (threadIdx.x == 0) {
    shared_val = max_val;
  }
  __syncthreads();
  max_val = shared_val;

  float sum = 0.0f;
  for (int i = tid; i < size; i += step) {
    x[i] = expf(x[i] - max_val);
    sum += x[i];
  }
  sum = BlockReduce(temp).Sum(sum);
  if (threadIdx.x == 0) {
    shared_val = sum;
  }
  __syncthreads();
  sum = shared_val;

  for (int i = tid; i < size; i += step) {
    x[i] /= sum;
  }
}

__global__ void multi_head_attention_kernel(int32_t pos, int32_t seq_len, float* query,
                                            float* score_ptr, float* output, float* key_cache,
                                            float* value_cache, int32_t kv_dim, int32_t kv_mul,
                                            int32_t head_num, int32_t head_size,
                                            int32_t layer_offset) {
  int head = blockIdx.x;
  if (head >= head_num) {
    return;
  }

  float* query_head = query + head * head_size;
  float* score_head = score_ptr + head * seq_len;
  float scale = 1.f / sqrtf(head_size);
  int32_t head_offset = (head / kv_mul) * head_size;
  for (int t = threadIdx.x; t <= pos; t += blockDim.x) {
    float* key_head = key_cache + layer_offset + t * kv_dim + head_offset;

    float score = 0.0f;
#pragma unroll
    for (int i = 0; i < head_size; i += 4) {
      float4 key_head_float4 = *reinterpret_cast<float4*>(key_head + i);
      float4 query_head_float4 = *reinterpret_cast<float4*>(query_head + i);
      if (i < head_size) {
        score += key_head_float4.x * query_head_float4.x;
      }
      if (i + 1 < head_size) {
        score += key_head_float4.y * query_head_float4.y;
      }
      if (i + 2 < head_size) {
        score += key_head_float4.z * query_head_float4.z;
      }
      if (i + 3 < head_size) {
        score += key_head_float4.w * query_head_float4.w;
      }
    }

    score *= scale;
    score_head[t] = score;
  }
  __syncthreads();

  softmax_gpu(score_head, pos + 1);
  __syncthreads();

  float* output_head = output + head * head_size;
  head_offset = layer_offset + (head / kv_mul) * head_size;
  for (int i = threadIdx.x; i < head_size; i += blockDim.x) {
    float value = 0.0f;
#pragma unroll
    for (int t = 0; t <= pos; t++) {
      float* value_head = value_cache + head_offset + t * kv_dim;
      float score = score_head[t];
      value += score * value_head[i];
    }
    output_head[i] = value;
  }
}

void mha_kernel_cu(int32_t pos, int32_t head_num, int32_t layer_index, int32_t seq_len,
                   int32_t kv_dim, int32_t kv_mul, int32_t head_size, const tensor::Tensor& mha_out,
                   const tensor::Tensor& query_tensor, const tensor::Tensor& score_tensor,
                   const tensor::Tensor& key_cache_tensor, const tensor::Tensor& value_cache_tensor,
                   base::DeviceType device_type, CudaConfig* config) {
  UNUSED(device_type);
  int32_t layer_offset = layer_index * seq_len * kv_dim;
  int32_t thread_num = 128;
  float* query = const_cast<float*>(query_tensor.ptr<float>());
  float* score = const_cast<float*>(score_tensor.ptr<float>());
  float* output = const_cast<float*>(mha_out.ptr<float>());

  float* key_cache = const_cast<float*>(key_cache_tensor.ptr<float>());
  float* value_cache = const_cast<float*>(value_cache_tensor.ptr<float>());

  hipStream_t stream = config->stream;
  multi_head_attention_kernel<<<head_num, thread_num, 0, stream>>>(
      pos, seq_len, query, score, output, key_cache, value_cache, kv_dim, kv_mul, head_num,
      head_size, layer_offset);
}

}  // namespace kernel